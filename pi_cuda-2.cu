#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define TRIALS_PER_THREAD 10000
#define BLOCKS 512
#define THREADS 1
#define PI 3.1415926535  // known value of pi
double uniform(double a, double b){
    return rand() / (RAND_MAX + 1.0) * (b - a) + a;
}
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	float x, y;

	hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


	for(int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}
double cpu_monte_carlo(long n) {
	double x, y;
	long in_circle;
	double r = 5.0;
	float a = -r,b = r;
	for(long i = 0; i < n; i++) {
		x = uniform(a,b);
		y = uniform(a,b);
		in_circle += x*x + y*y <= r*r ? 1 : 0;
	}
	return ((double)in_circle/n)*4;
}
int main (int argc, char *argv[]) {
	clock_t startgpu, stopgpu,startcpu,stopcpu;
	float host[BLOCKS * THREADS];
	float *dev;
	hiprandState *devStates;

	startcpu = clock();
	float pi_cpu = cpu_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD);
	stopcpu = clock();
	printf("Pi = %f	CPU pi calculated in %f s.\n", pi_cpu,(stopcpu-startcpu)/(float)CLOCKS_PER_SEC);


	startgpu = clock();

	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(float)); // allocate device mem. for counts
	
	hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );

	gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost); // return results 

	float pi_gpu;
	for(int i = 0; i < BLOCKS * THREADS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * THREADS);

	stopgpu = clock();
	printf("Pi = %f	GPU pi calculated in %f s.\n", pi_gpu,(stopgpu-startgpu)/(float)CLOCKS_PER_SEC);

	
	return 0;
}