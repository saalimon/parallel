#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#define ROUNDS 1000000
#define BLOCKS 512
#define GRIDS 1
double uniform(double a, double b){
    return rand() / (RAND_MAX + 1.0) * (b - a) + a;
}
__global__ void gpu_monte_carlo(float *pi, hiprandState *states) {
	unsigned int t_rank = threadIdx.x + blockDim.x * blockIdx.x;
	long in_circle = 0;
	float x, y;

	hiprand_init(1234, t_rank, 0, &states[t_rank]);  // 	Initialize CURAND


	for(long i = 0; i < ROUNDS; i++) {
		x = hiprand_uniform (&states[t_rank]);
		y = hiprand_uniform (&states[t_rank]);
		in_circle += x*x + y*y <= 1.0f ? 1 : 0; // count if x & y is in the circle.
	}
	pi[t_rank] = (float) in_circle / ROUNDS * 4.0; // return estimate of pi
}
double cpu_monte_carlo(long n) {
	double x, y;
	long in_circle;
	double r = 5.0;
	float a = -r,b = r;
	for(long i = 0; i < n; i++) {
		x = uniform(a,b);
		y = uniform(a,b);
		in_circle += x*x + y*y <= r*r ? 1 : 0;
	}
	return (double) in_circle / n * 4;
}
int main (int argc, char *argv[]) {
	clock_t startgpu, stopgpu,startcpu,stopcpu;
	float host[BLOCKS * GRIDS];
	float *dev;
	hiprandState *devStates;

	startcpu = clock();
	float pi_cpu = cpu_monte_carlo(BLOCKS * GRIDS * ROUNDS);
	stopcpu = clock();
	printf("Pi = %f	CPU pi calculated in %f s.\n", pi_cpu,(stopcpu-startcpu)/(float)CLOCKS_PER_SEC);


	startgpu = clock();

	hipMalloc((void **) &dev, BLOCKS * GRIDS * sizeof(float)); // allocate device mem. for counts
	
	hipMalloc( (void **)&devStates, GRIDS * BLOCKS * sizeof(hiprandState) );

	gpu_monte_carlo<<<BLOCKS, GRIDS>>>(dev,devStates);

	hipMemcpy(host, dev, BLOCKS * GRIDS * sizeof(float), hipMemcpyDeviceToHost); // return results 

	float pi_gpu;

	for(int i = 0; i < BLOCKS * GRIDS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * GRIDS);

	stopgpu = clock();
	printf("Pi = %f	GPU pi calculated in %f s.\n", pi_gpu,(stopgpu-startgpu)/(float)CLOCKS_PER_SEC);

	
	return 0;
}