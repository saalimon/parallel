#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>     /* srand, rand */
#include<time.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#define grid_size 1
#define block_size 512
#define n 1000000
//rand() in gpu
__global__ void cal_pi(double *sum_D)
{
    double x,y;
    int i;
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    int points_in_circle = 0;
    

    hiprandState_t rng;
    hiprand_init(clock64(),tid,0,&rng);

    for(i=0;i<n;i++){
        x = hiprand_uniform(&rng);
        y = hiprand_uniform(&rng);
        points_in_circle += (x*x + y*y <= 1.0f);
    }
    sum_D[tid] += 4.0f * points_in_circle / (float) n;
}
double uniform(double a, double b){
    return rand() / (RAND_MAX + 1.0) * (b - a) + a;
}
double cpu_monte_carlo(long n){
	double x, y;
	long in_circle = 0,i;
	double r = 5.0;
	float a = -r,b = r;
	for(i = 0; i < n; i++) {
		x = uniform(a,b);
		y = uniform(a,b);
		in_circle += x*x + y*y <= r*r ? 1 : 0;
	}
	return ((double) in_circle / n * 4);
}
int main(int argc, char **argv){
    int i;
    double pi_gpu,size;
    double *sum_h, *sum_D;
    clock_t start_gpu, end_gpu;
    clock_t start_cpu, end_cpu;
    
    size = grid_size*block_size*sizeof(double);
    sum_h = (double *)malloc(size);
    //start run in cpu
    start_cpu = clock();
    double pi_cpu = cpu_monte_carlo(block_size * grid_size * n);
    end_cpu = clock();
    printf("Pi by CPU = %lf from ramdom %d dots\n",pi_cpu,n);
    printf("Time: %f sec.\n",(end_cpu - start_cpu)/(float)CLOCKS_PER_SEC);

    //start run in gpu
    start_gpu = clock();
    hipMalloc((void **)&sum_D, size);
    cal_pi<<<block_size, grid_size>>>(sum_D);
    hipMemcpy(sum_h, sum_D, size,hipMemcpyDeviceToHost);
    pi_gpu = 0;
    for(i = 0; i< grid_size*block_size;i++){
        pi_gpu += sum_h[i];
    }
    pi_gpu /=(block_size*grid_size);
    end_gpu = clock();
    
    printf("Pi by GPU = %lf from ramdom %d dots\n",pi_gpu,n);
    printf("Time: %f sec.\n",(end_gpu - start_gpu)/(float)CLOCKS_PER_SEC);
    hipFree(sum_D);
    free(sum_h);

    return 0;
}