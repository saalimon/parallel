
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matAdd(double *a_D, double *c_D)
{
   int t_rank;

   t_rank = threadIdx.y*blockDim.x + threadIdx.x;
   c_D[t_rank] = t_rank+1;
}

int main(int argc, char **argv)
{
   int i, j;
   int size, block_size = 8, grid_size = 1;
   double *c_H, *c_D, *a_H, *a_D;
   
   size = block_size*block_size*sizeof(double);
   a_H = (double *)malloc(size);
   c_H = (double *)malloc(size);
   hipMalloc((void **)&a_D, size);
   hipMalloc((void **)&c_D, size);
   
   for (i=0; i<block_size; i++)
      for (j=0; j<block_size; j++)
         a_H[(i*block_size)+j]=(i*block_size)+j;
   for(i = 0; i < block_size; i++, printf("\n"))
      for (j=0; j < block_size; j++) 
         //printf("matric[%d][%d] is %f\n", i, j, c_H[(i*block_size)+j]);
         printf(" %4.1f", a_H[(i*block_size)+j]);
   printf("\n");
   hipMemcpy(a_D, a_H, size, hipMemcpyHostToDevice);

   dim3 Block(block_size, block_size);
   dim3 Grid(grid_size, grid_size);

   matAdd<<<Grid, Block>>>(a_D, c_D);

   hipMemcpy(c_H, c_D, size, hipMemcpyDeviceToHost);

   for(i = 0; i < block_size; i++, printf("\n"))
      for (j=0; j < block_size; j++) 
         //printf("matric[%d][%d] is %f\n", i, j, c_H[(i*block_size)+j]);
         printf(" %4.1f", c_H[(i*block_size)+j]);

   free(a_H);
   free(c_H);
   hipFree(a_D);
   hipFree(c_D);

   return 0;
}

